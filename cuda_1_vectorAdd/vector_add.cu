#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>

__global__ void add(int *a, int *b, int *c, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    { 
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char* argv[])
{

    if (argc != 4)
    {
        fprintf(stderr, "You have to provide ARRAY_SIZE, NUM_BLOCK, BLOCK_SIZE and  as arguments.\n");
        return -1;
    }
    
    char *p;

    int ARRAY_SIZE;
    int ul_conv = strtoul(argv[1], &p, 10);
    ARRAY_SIZE = ul_conv;

    int NUM_BLOCK;
    ul_conv = strtoul(argv[2], &p, 10);
    NUM_BLOCK = ul_conv;

    int BLOCK_SIZE;
    ul_conv = strtoul(argv[3], &p, 10);
    BLOCK_SIZE = ul_conv;

    int N = ARRAY_SIZE;
    int a[N], b[N], c[N];

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);
   
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add<<<NUM_BLOCK, BLOCK_SIZE>>>(dev_a, dev_b, dev_c, N);

    hipDeviceSynchronize();
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    // cleanup
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf ("%d, %d, %d, %f\n", ARRAY_SIZE, NUM_BLOCK, BLOCK_SIZE, time);
    return 0;
}
