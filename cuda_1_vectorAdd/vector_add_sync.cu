#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>

void add(int* a, int* b, int* c, int N)
{ 
    int i;
    for (i = 0; i < N; i ++)
    { 
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char* argv[])
{
    if (argc != 2)
    {
        fprintf(stderr, "You have to provide ARRAY_SIZE as argument.\n");
        return -1;
    }

    char *p;

    int ARRAY_SIZE;
    int ul_conv = strtoul(argv[1], &p, 10);
    ARRAY_SIZE = ul_conv;

    int N = ARRAY_SIZE;
    int* a = (int *)malloc(N * sizeof(int));
    int* b = (int *)malloc(N * sizeof(int));
    int* c = (int *)malloc(N * sizeof(int));

    int i;
    for (i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

   
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add(a, b, c, N);

    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);


    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", a[i], b[i], c[i]);
 
    //}

    // cleanup
    free(a);
    free(b);
    free(c);
     
    printf ("%d, %f\n", ARRAY_SIZE, time);
    return 0;
}
