#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <helper_timer.h>
#define BLOCK_SIZE 16

typedef struct
{	int WIDTH;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.WIDTH = A.WIDTH; d_A.height = A.height;
	size_t size = A.WIDTH * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.WIDTH = B.WIDTH; d_B.height = B.height;
	size = B.WIDTH * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.WIDTH = C.WIDTH; d_C.height = C.height;
	size = d_C.WIDTH * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);
	
	// Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.WIDTH / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// each thread computes one element of C and acumulates results to Cvalue
	 float Cvalue = 0;
	 int row = blockIdx.y * blockDim.y + threadIdx.y;
	 int col = blockIdx.x * blockDim.x + threadIdx.x;
	if ((row>=A.height) || (col>=B.WIDTH))
	{
		return;
	}
	for (int e=0; e<A.WIDTH; e++){
		Cvalue += A.elements[row*A.WIDTH + e] * B.elements[e*B.WIDTH + col];
		C.elements[row*C.WIDTH + col] = Cvalue;
	}
}

//square matrix multiplication cpu
void MatMulCPU(Matrix A, Matrix B, Matrix C)
{
	int i,j,k, n = A.WIDTH;
	for(i = 0; i < n; ++i){
		for(j = 0; j < n; ++j)
		{
			C.elements[i*C.WIDTH +j] = 0;
			for(k = 0; k < n; ++k)
            {
                C.elements[i*C.WIDTH +j] += A.elements[i*A.WIDTH +k] * B.elements[k*A.WIDTH +j];
            }
		} 
	}
		        
}

int main(int argc, char * const argv[])
{	
	if (argc != 4)
    {
        fprintf(stderr, "You have to provide WIDTH, ARRAY_A, ARRAY_B  as arguments.\n");
        return -1;
    }
    
    char *p;

    int WIDTH;
    WIDTH = strtoul(argv[1], &p, 10);

    char* ARRAY_A;
    ARRAY_A = argv[2];

    char* ARRAY_B;
	ARRAY_B = argv[3];
	
	StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
		
	Matrix A;
	Matrix B;
	Matrix C_gpu;
	Matrix C_cpu;
	
	A.WIDTH = WIDTH;
	B.WIDTH = WIDTH;
	C_gpu.WIDTH = WIDTH;
	C_cpu.WIDTH = WIDTH;
	
	A.height = WIDTH;
	B.height = WIDTH;
	C_gpu.height = WIDTH;
	C_cpu.height = WIDTH;
	
	A.elements = new float[WIDTH*WIDTH];
	B.elements = new float[WIDTH*WIDTH];
	C_gpu.elements = new float[WIDTH*WIDTH];
	C_cpu.elements = new float[WIDTH*WIDTH];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open(ARRAY_A);
	B_input.open(ARRAY_B);
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;
	while (!A_input.eof())
	{	A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}
	A_input.close();
	B_input.close();

	sdkResetTimer(&timer);
        sdkStartTimer(&timer);
	MatMul(A, B, C_gpu);
        sdkStopTimer(&timer);
	float time_gpu = sdkGetTimerValue(&timer);
	
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	MatMulCPU(A, B, C_cpu);
	float time_cpu = sdkGetTimerValue(&timer);

	//check if the same
	for (int i=0; i<WIDTH; i++)
	{	for (int j=0; j<WIDTH; j++)
			if (C_gpu.elements[i*WIDTH+j] != C_cpu.elements[i*WIDTH+j]){
				printf("something is no yes");
			}
	}

	std::ofstream C_output;
	C_output.open("C.txt");
	for (int i=0; i<WIDTH; i++)
	{	for (int j=0; j<WIDTH; j++)
			C_output<<C_cpu.elements[i*WIDTH+j]<<"\t";
		C_output<<endl;
	}

	printf ("%d, %f, %f\n", WIDTH, time_gpu, time_cpu);
	sdkDeleteTimer(&timer);
}	
