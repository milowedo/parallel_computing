// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#define BLOCK_SIZE 16

typedef struct
{	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // define the block size (what is the best value?) 
        dim3 dimGrid(128); //  choose grid size depending on problem size 
        
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// each thread computes one element of C and acumulates results to Cvalue
	 float Cvalue = 0;
	 int row = blockIdx.y * blockDim.y + threadIdx.y;
	 int col = blockIdx.x * blockDim.x + threadIdx.x;
	if ((row>=A.height) || (col>=B.width))
	{
		return;
	}
	for (int e=0; e<A.width; e++){
		Cvalue += A.elements[row*A.width + e] * B.elements[e*B.width + col];
		C.elements[row*C.width + col] = Cvalue;
	}
}

//square matrix multiplication cpu
void MatMulCPU(Matrix A, Matrix B, Matrix C)
{
	int i,j,k, n = A.width;
	for(i = 0; i < n-1; ++i){
		for(j = 0; j < n-1; ++j)
		{
			C.elements[i*C.width +j] = 0;
			for(k = 0; k < n-1; ++k)
            {
                C.elements[i*C.width +j] += A.elements[i*A.width +k] * B.elements[k*A.width +j];
            }
		} 
	}
		        
}

int main(int argc, char * const argv[])
{	
	int Width = 16;
	
	Matrix A;
	Matrix B;
	Matrix C_gpu;
	Matrix C_cpu;
	
	A.width = Width;
	B.width = Width;
	C_gpu.width = Width;
	C_cpu.width = Width;
	
	A.height = Width;
	B.height = Width;
	C_gpu.height = Width;
	C_cpu.height = Width;
	
	A.elements = new float[Width*Width];
	B.elements = new float[Width*Width];
	C_gpu.elements = new float[Width*Width];
	C_cpu.elements = new float[Width*Width];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open("A.txt");
	B_input.open("B.txt");
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;
	while (!A_input.eof())
	{	A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}
	A_input.close();
	B_input.close();

	MatMul(A, B, C_gpu);
	MatMulCPU(A, B, C_cpu);

	//check if the same
	for (int i=0; i<Width; i++)
	{	for (int j=0; j<Width; j++)
			if (C_gpu.elements[i*Width+j] != C_cpu.elements[i*Width+j]){
				std::cout << "Different results.";
			}
		std::cout << "Same results.";
	}

	std::ofstream C_output;
	C_output.open("C.txt");
	for (int i=0; i<Width; i++)
	{	for (int j=0; j<Width; j++)
			C_output<<C_gpu.elements[i*Width+j]<<"\t";
		C_output<<endl;
	}

}	
