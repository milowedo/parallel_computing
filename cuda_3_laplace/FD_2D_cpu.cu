#include "hip/hip_runtime.h"
/*** Calculating a derivative with CD ***/
#include <iostream>
#include <fstream>
#include <cmath>
#include <sys/time.h>

void copy_array(float *u, float *u_prev, int N)
{
    int i;
    for(i = 0; i< N*N; i++){
        u_prev[i] = u[i];
    }
}

void update (float *u, float *u_prev, int N, float h, float dt, float alpha)
{
	// Setting up indices
	int I = 0;
    
    for(; I < N*N; I++){
        if ( (I>N) && (I< N*N-1-N) && (I%N!=0) && (I%N!=N-1)) 
        {	
            u[I] = u_prev[I] + alpha*dt/(h*h) * (u_prev[I+1] + u_prev[I-1] + u_prev[I+N] + u_prev[I-N] - 4*u_prev[I]);
        }
    }
	
	// Boundary conditions are automatically imposed
	// as we don't touch boundaries
}

double get_time()
{
  struct timeval tim; 
  hipDeviceSynchronize(); 
  gettimeofday(&tim, NULL); 
  return (double) tim.tv_sec+(tim.tv_usec/1000000.0); 
}

int main(int argc, char * const argv[])
{
    int N;		// For textures to work, N needs to be a multiple of
	int BLOCKSIZE;	// 32. As I will be using BLOCKSIZE to be a multiple of 8
						// I'll just look for the closest multiple of BLOCKSIZE (N_max)

	if (argc != 2)
    {
        fprintf(stderr, "You have to provide size(n)  as arguments.\n");
        return -1;
	}

	char *p;

	N = strtoul(argv[1], &p, 10);

	float xmin 	= 0.0f;
	float xmax 	= 3.5f;
	float ymin 	= 0.0f;
	//float ymax 	= 2.0f;
	float h   	= (xmax-xmin)/(N-1);
	float dt	= 0.00001f;	
	float alpha	= 0.645f;
	float time 	= 0.4f;

	int steps = ceil(time/dt);
	int I;

	float *x  	= new float[N*N]; 
	float *y  	= new float[N*N]; 
	float *u  	= new float[N*N];
	float *u_prev  	= new float[N*N];


	// Generate mesh and intial condition
	for (int j=0; j<N; j++)
	{	for (int i=0; i<N; i++)
		{	I = N*j + i;
			x[I] = xmin + h*i;
			y[I] = ymin + h*j;
			u[I] = 0.0f;
			if ( (i==0) || (j==0)) 
				{u[I] = 200.0f;}
		}
	}

	
	// Loop 
	double start = get_time();
	for (int t=0; t<steps; t++)
	{	copy_array(u, u_prev, N);
		update(u, u_prev, N, h, dt, alpha);

	}
	double stop = get_time();
	
	double elapsed = stop - start;
	printf("%d, 0, %f\n", N, elapsed);

	// std::ofstream temperature("temperature_cpu.txt");
	// for (int j=0; j<N; j++)
	// {	for (int i=0; i<N; i++)
	// 	{	I = N*j + i;
	// 	//	std::cout<<u[I]<<"\t";
	// 		temperature<<x[I]<<"\t"<<y[I]<<"\t"<<u[I]<<std::endl;
	// 	}
	// 	temperature<<"\n";
	// 	//std::cout<<std::endl;
	// }

	// temperature.close();
}
