#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>

__global__ void add(int *a, int *b, int *c, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    { 
        c[i] = a[i] + b[i];
    }
}

int main(void)
{
    int N = 10;
    int a[N], b[N], c[N];

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);
   
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add<<<1, 1>>>(dev_a, dev_b, dev_c, N);

    hipDeviceSynchronize();
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    // cleanup
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf ("Time for the kernel: %f ms\n", time);
    return 0;
}
