#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>

__global__ void add(int *a, int *b, int *c, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    { 
        c[i] = a[i] + b[i];
    }
}

int main(void)
{

    if (argc != 4)
    {
        fprintf(stderr, "You have to provide ARRAY_SIZE, THREAD_COUNT, BLOCK_COUNT and  as arguments.\n");
        return -1;
    }
    
    char *p;

    int ARRAY_SIZE;
    ul_conv = strtoul(argv[1], &p, 10);
    ARRAY_SIZE = ul_conv;

    int THREAD_COUNT;
    ul_conv = strtoul(argv[2], &p, 10);
    THREAD_COUNT = ul_conv;

    int BLOCK_COUNT;
    ul_conv = strtoul(argv[3], &p, 10);
    BLOCK_COUNT = ul_conv;
    
    int N = 10;
    int a[N], b[N], c[N];

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);
   
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add<<<1, 1>>>(dev_a, dev_b, dev_c, N);

    hipDeviceSynchronize();
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    // cleanup
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf ("Time for the kernel: %f ms\n", time);
    return 0;
}
