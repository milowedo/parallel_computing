#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>

void add(int* a, int* b, int* c, int N)
{ 
    int i;
    for (i = 0; i < N; i ++)
    { 
        c[i] = a[i] + b[i];
    }
}

int main(void)
{
    if (argc != 4)
    {
        fprintf(stderr, "You have to provide ARRAY_SIZE, THREAD_COUNT, BLOCK_COUNT and  as arguments.\n");
        return -1;
    }
    
    char *p;

    int ARRAY_SIZE;
    ul_conv = strtoul(argv[1], &p, 10);
    ARRAY_SIZE = ul_conv;

    int THREAD_COUNT;
    ul_conv = strtoul(argv[2], &p, 10);
    THREAD_COUNT = ul_conv;

    int BLOCK_COUNT;
    ul_conv = strtoul(argv[3], &p, 10);
    BLOCK_COUNT = ul_conv;

    int N = 10;
    int* a = (int *)malloc(N * sizeof(int));
    int* b = (int *)malloc(N * sizeof(int));
    int* c = (int *)malloc(N * sizeof(int));

    int i;
    for (i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

   
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add(a, b, c, N);

    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);


    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", a[i], b[i], c[i]);
 
    //}

    // cleanup
    free(a);
    free(b);
    free(c);
     
    printf ("Time for the kernel: %f ms\n", time);
    return 0;
}
