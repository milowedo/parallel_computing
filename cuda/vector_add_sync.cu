#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>

void add(int* a, int* b, int* c, int N)
{ 
    int i;
    for (i = 0; i < N; i ++)
    { 
        c[i] = a[i] + b[i];
    }
}

int main(void)
{
    int N = 10;
    int* a = (int *)malloc(N * sizeof(int));
    int* b = (int *)malloc(N * sizeof(int));
    int* c = (int *)malloc(N * sizeof(int));

    int i;
    for (i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

   
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add(a, b, c, N);

    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);


    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", a[i], b[i], c[i]);
 
    //}

    // cleanup
    free(a);
    free(b);
    free(c);
     
    printf ("Time for the kernel: %f ms\n", time);
    return 0;
}
